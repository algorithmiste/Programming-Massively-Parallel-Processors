#include "hip/hip_runtime.h"


#include <stdio.h>
#include <math.h>


#define PRINT_ERROR(err) {\
	if (err != hipSuccess) {\
		printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__ );\
		exit(EXIT_FAILURE);\
	}\
};

/* ERROR CHECKING AND HANDLING IN CUDA:
	It is important for a program to check and handle errors.
	CUDA API functions return flags that indicate whether an error has
		occurred when they served theh request. Most errors are due to 
		inappropriate argument values used in the call. See below examples.*/

		// Compute vector sum C = A+B
		// Each thread performs one pair-wise addition
__global__
void vecAddKernel(float* A, float* B, float* C, int n) {

	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < n)
		C[i] = A[i] + B[i];

}

void vecAdd(float* h_A, float* h_B, float* h_C, int n) {
	int size = n * sizeof(float);
	float *d_A, *d_B, *d_C;
	// 1. Allocate device memory for A, B, and C
	//	  copy A and B to device memory
	hipError_t err = hipMalloc((void**)& d_A, size);
	PRINT_ERROR(err);

	err = hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
	PRINT_ERROR(err);

	err = hipMalloc((void**)& d_B, size);
	PRINT_ERROR(err);
	err = hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);
	PRINT_ERROR(err);
	err = hipMalloc((void**)& d_C, size);
	PRINT_ERROR(err);
	// 2. Kernel launch code - to have the device perform the actual vector addition
	int nBlocks = ceil(n / 256.0);
	vecAddKernel<<<nBlocks, 256>>>(d_A, d_B, d_C, n);
	hipDeviceSynchronize();
	
	// 3. copy C from the device memory 
	//	  free device vectors
	err = hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);
	PRINT_ERROR(err);

	err = hipFree(d_A);
	PRINT_ERROR(err);

	err = hipFree(d_B);
	PRINT_ERROR(err);

	err = hipFree(d_C);
	PRINT_ERROR(err);
}

